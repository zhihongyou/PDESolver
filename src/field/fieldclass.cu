#include "hip/hip_runtime.h"
#ifndef FIELDCLASS_CU
#define FIELDCLASS_CU

#include <iostream> 
#include <vector>
#include <fstream>
#include <random>
#include <math.h>
#include <cmath>
#include <sstream>
#include <iomanip>
#include "fieldclass.h"
#include "fieldclassGPU.cu"
#include "fieldclass_fieldfunctionsGPU.cu"
#include "fieldclass_boundaryconditionGPU.cu"
#include "fieldclass_initialconditionGPU.cu"
#include "fieldclass_fieldfunctions.cu"
#include "fieldclass_boundarycondition.cu"
#include "fieldclass_initialcondition.cu"


using namespace std;


// =======================================================================
// Constructors
Field::Field (Mesh* mesh_ptr_t, string name_t) {
    traits_host.mesh_ptr=mesh_ptr_t;
    traits_host.name=name_t;
    traits_host.rank=1;
    traits_host.priority=0;
    traits_host.boun_cond="periodic";
    traits_host.init_cond="sin";
    traits_host.expo_data="on";
    // Initiate field on host, which will then be copied to f.
    allocField<double>(f_host[0], "cpu");
    if (traits_host.init_cond=="Gaussian") {
        initFieldGaus(0,10,1);
    } else if (traits_host.init_cond=="ones") {
        initFieldConst(1);
    } else if (traits_host.init_cond=="sin") {
        initFieldSin(0.01,4,0);        
    };
    num_f_funcs=0;
    for (int i=0; i<200; i++) {
        f_funcs_host[i]=NULL;
    };
};


// =======================================================================
// Constructors
Field::Field (Mesh* mesh_ptr_t, string name_t, int priority_t) {
    traits_host.mesh_ptr=mesh_ptr_t;
    traits_host.name=name_t;
    traits_host.rank=1;
    traits_host.priority=priority_t;
    traits_host.boun_cond="periodic";
    traits_host.init_cond="sin";
    traits_host.expo_data="on";
    // Initiate field on host, which will then be copied to f.
    allocField<double>(f_host[0], "cpu");
    if (traits_host.init_cond=="Gaussian") {
        initFieldGaus(0,10,1);
    } else if (traits_host.init_cond=="ones") {
        initFieldConst(1);
    } else if (traits_host.init_cond=="sin") {
        initFieldSin(0.01,4,0);        
    };
    num_f_funcs=0;
    for (int i=0; i<200; i++) {
        f_funcs_host[i]=NULL;
    };
};


// -----------------------------------------------------------------------
Field::Field (Mesh* mesh_ptr_t, string name_t, int rank_t, int priority_t, string boun_cond_t, string init_cond_t, string expo_data_t) {
    traits_host.mesh_ptr=mesh_ptr_t;
    traits_host.name=name_t;
    traits_host.rank=rank_t;
    traits_host.priority=priority_t;
    traits_host.boun_cond=boun_cond_t;
    traits_host.init_cond=init_cond_t;
    traits_host.expo_data=expo_data_t;
    // Initiate field on host, which will then be copied to f.
    allocField<double>(f_host[0], "cpu");
    if (traits_host.init_cond=="Gaussian") {
        initFieldGaus(0,10,1);
    } else if (traits_host.init_cond=="ones") {
        initFieldConst(1);
    } else if (traits_host.init_cond=="sin") {
        initFieldSin(0.01,4,0);        
    };
    num_f_funcs=0;
    for (int i=0; i<200; i++) {
        f_funcs_host[i]=NULL;
    };
};


// ----------------------------------------------------------------------
void Field::setFieldConstCPU(double* f_t, double f_val, int Nx, int Ny, int Nbx, int Nby) {
    setFieldConstCPUCore(f_t, f_val, Nx, Ny, Nbx, Nby);
};


// ----------------------------------------------------------------------
void Field::setFieldConstGPU(double* f_t, double f_val, int Nx, int Ny, int Nbx, int Nby) {
    setFieldConstGPUCore<<<Ny,Nx>>>(f_t, f_val, Nx, Ny, Nbx, Nby);
};


// ----------------------------------------------------------------------
void Field::setRhsTerms(vector<rhsTerm> rhs_terms_t) {
    rhs_terms=rhs_terms_t;
    FFuncArgs f_func_args1={gridNumber().x,gridNumber().y,gridNumberBoun().x,gridNumberBoun().y,gridSize().x,gridSize().y};
    for (int i=0; i<rhs_terms.size(); i++) {        
        for (int j=0; j<rhs_terms[i].f_funcs.size(); j++) {
            rhs_terms[i].f_funcs[j].f_func_args=f_func_args1;
        };
    };
};


// ----------------------------------------------------------------------
void Field::export_conf(string str_t, string device, int include_boun=0) {
    if (device=="cpu") {        
        export_conf_any(f[0],name(),str_t, "cpu", include_boun);
    } else if (device=="gpu") {
        export_conf_any(f[0],name(),str_t, "gpu", include_boun);
    };
}

// ----------------------------------------------------------------------
void Field::export_f_func(string f_operator, string str_t, string device, int include_boun=0) {

    int f_func_idx;
    for (int i=0; i<num_f_funcs; i++) {
        if (f_funcs_rhs[i].f_operator==f_operator) {
            f_func_idx=f_funcs_rhs[i].f_func_idx;
        };
    };
    
    if (device=="cpu") {        
        export_conf_any(f_funcs_host[f_func_idx], name()+"_"+f_operator, str_t, "cpu", include_boun);
    } else if (device=="gpu") {
        export_conf_any(f_funcs_host[f_func_idx], name()+"_"+f_operator, str_t, "gpu", include_boun);
    };
}


// ----------------------------------------------------------------------
void Field::export_conf_any(double* f_t, string f_name, string str_t, string location_t="cpu" , int include_boun=0) {
    ofstream conf_file;
    int PrecData=8;
    string conf_file_name="data/"+f_name+"_"+ str_t + ".dat";
    conf_file.open(conf_file_name.c_str() );
    
    int idx;
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    int* idx0=new int [4];

    
    if (location_t=="gpu") {
        allocField<double>(f_temp_host, "cpu");
        hipMemcpy(f_temp_host, f_t, gridNumberAll()*sizeof(double),hipMemcpyDeviceToHost);
    };        
    
    if (include_boun==0) {
        idx0[0]=0;
        idx0[1]=0;
        idx0[2]=0;
        idx0[3]=0;
    } else {
        idx0[0]=-Nbx;
        idx0[1]=Nbx;
        idx0[2]=-Nby;
        idx0[3]=Nby;
    };

    for (int j=idx0[2]; j<Ny+idx0[3]; j++) {
        for (int i=idx0[0]; i<Nx+idx0[1]; i++) {        
            idx=(Nx+2*Nbx)*(j+Nby)+i+Nbx;
            if (location_t=="cpu") {
                conf_file<<setiosflags(ios::scientific) <<setprecision(PrecData) <<f_t[idx]<<endl;
            } else {
                conf_file<<setiosflags(ios::scientific) <<setprecision(PrecData) <<f_temp_host[idx]<<endl;
            };
        }
    }
    conf_file.close();
}

// ----------------------------------------------------------------------
// string Field::equation() {
//     string eqn;
//     if (priority==0) {
//         eqn="p_t "+name()+"=";
//     } else {
//         eqn=name()+"=";
//     };
//     for (auto rhs_term_i : rhsTerms()) {
//         if (rhs_term_i != rhsTerms().begin()) {
            
//         };
//         for (auto f_func_i : rhs_term_i.f_function) {
//             eqn=eqn+
//                 cout<<"*"<<f_func_i.f_operator<<"("<<(*f_func_i.field_ptr).name() <<")";
//                 evalOperator(f_ptr_i,f_func_i,f_func_ptrs[N_funcs],i_field);
//                 N_funcs+=1;
//             };
//             addRHSTerm(f_ptr_i,i_field,rhs_term_i,f_func_ptrs,N_funcs);
//         };

//         if ((*f_ptr_i).priority()>0 && (*f_ptr_i).bounCond()=="periodic") {
//             if (device=="cpu") {
//                 (*f_ptr_i).applyBounCondPeriCPU(i_field);
//             } else if (device=="gpu"){
//                 (*f_ptr_i).applyBounCondPeriGPU(i_field);
//             };
//         };
// };

// -------------------------------------------------------------------
// Copy any field data from CPU to GPU
void Field::updateAnyFieldDev (double* f_dev_ptr, double * f_host_ptr) {

    allocField<double>(f_dev_ptr, "gpu");
    hipMemcpy(f_dev_ptr, f_host_ptr, gridNumberAll()*sizeof(double),hipMemcpyHostToDevice);
};

// -------------------------------------------------------------------
// Copy any field data from GPU to CPU
void Field::updateAnyFieldHost (double* f_host_ptr, double * f_dev_ptr) {    
    hipMemcpy(f_host_ptr, f_dev_ptr, gridNumberAll()*sizeof(double),hipMemcpyDeviceToHost);
};

// -------------------------------------------------------------------
// Copy main field data from CPU to GPU
void Field::updateMainFieldDev () {
    allocField<double>(f[0], "gpu");
    hipMemcpy(f[0], f_host[0], gridNumberAll()*sizeof(double),hipMemcpyHostToDevice);
};

// -------------------------------------------------------------------
// Copy main field data from GPU to CPU
void Field::updateMainFieldHost () {
    allocField<double>(f[0], "cpu");
    hipMemcpy(f_host[0], f[0], gridNumberAll()*sizeof(double),hipMemcpyDeviceToHost);
};

// ------------------------------------------------------------------
template <typename T>
void Field::allocField (T* &f_t, string location) {
    if (f_t==NULL) {
        if (location=="cpu") {
            f_t=new T[gridNumberAll()];
        } else if (location=="gpu") {
            hipMalloc(&f_t, gridNumberAll()*sizeof(T));
        };
    };    
};

// ----------------------------------------------------------------------
double* Field::getFFuncPtr(string f_operator) {

    double* f_func_ptr;
    for (int i=0; i<num_f_funcs; i++) {
        if (f_funcs_rhs[i].f_operator==f_operator) {
            f_func_ptr=f_funcs_host[f_funcs_rhs[i].f_func_idx];
        };
    };

    return f_func_ptr;
        
};


// -----------------------------------------------------------------------
void Field::addFunctoRHS(FFuncDef f_func_i, string device, string func_scheme) {
    FFuncItem f_func_item;
    f_func_item.f_operator=f_func_i.f_operator;
    f_func_item.f_func_idx=num_f_funcs;
    f_func_item.f_func_args=f_func_i.f_func_args;
    
    if (device=="cpu") {
        allocField<double>(f_funcs_host[num_f_funcs], "cpu");
        if (f_func_map_all[{f_func_i.f_operator,func_scheme}]==0) {
            f_func_item.f_func=f_func_map_all[{f_func_i.f_operator,""}];
        } else {
            f_func_item.f_func=f_func_map_all[{f_func_i.f_operator,func_scheme}];
        };
    } else if (device=="gpu") {
        allocField<double>(f_funcs_host[num_f_funcs], "gpu");
        if (f_func_map_all[{f_func_i.f_operator,func_scheme}]==0) {
            f_func_item.f_func=f_func_map_all_dev[{f_func_i.f_operator,""}];
        } else {
            f_func_item.f_func=f_func_map_all_dev[{f_func_i.f_operator,func_scheme}];
        };
    };
    f_funcs_rhs[num_f_funcs]=f_func_item;
    num_f_funcs+=1;
};



// ------------------------------------------------------------------


// =================================================================

#endif
