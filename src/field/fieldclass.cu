#include "hip/hip_runtime.h"
#ifndef FIELDCLASS_CU
#define FIELDCLASS_CU

#include <iostream> 
#include <vector>
#include <fstream>
#include <random>
#include <math.h>
#include <cmath>
#include <sstream>
#include <iomanip>
#include "fieldclass.h"
#include "fieldclassGPU.cu"


using namespace std;


// =======================================================================
// Constructor
Field::Field (Mesh* mesh_ptr_t, string name_t, int rank_t, int priority_t, string boun_cond_t, string init_cond_t, string expo_data_t) {
    traits_host.mesh_ptr=mesh_ptr_t;
    traits_host.name=name_t;
    traits_host.rank=rank_t;
    traits_host.priority=priority_t;
    traits_host.boun_cond=boun_cond_t;
    traits_host.init_cond=init_cond_t;
    traits_host.expo_data=expo_data_t;
    // Initiate field on host, which will then be copied to f.
    allocField(f_host[0], "cpu");
    if (traits_host.init_cond=="Gaussian") {
        initFieldGaus(0,10,1);
    } else if (traits_host.init_cond=="ones") {
        initFieldConst(1);
    } else if (traits_host.init_cond=="sin") {
        initFieldSin(0.01,4,0);        
    };
};

// -----------------------------------------------------------------------
void Field::initFieldConst(double f_value) {    
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    for (int j=0; j<Ny; j++) {
        for (int i=0; i<Nx; i++) {
            int idx=(j+Nby)*(Nx+2*Nbx)+i+Nbx;
            f_host[0][idx]=f_value;
        };
    };
    applyBounCondPeriCPU(f_host[0]);
};

// -----------------------------------------------------------------------
void Field::initFieldGaus(double r_center, double r_decay, double gaus_amplitude) {
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    double dx2=gridSize().x*gridSize().x;
    double dy2=gridSize().y*gridSize().y;
    double rd2=r_decay*r_decay;
    for (int j=0; j<Ny; j++) {
        for (int i=0; i<Nx; i++) {
            int idx=(j+Nby)*(Nx+2*Nbx)+i+Nbx;
            double r2=dx2*(i-0.5*Nx)*(i-0.5*Nx)+dy2*(j-0.5*Ny)*(j-0.5*Ny);
            f_host[0][idx]=gaus_amplitude*(exp(-r2/rd2));
        };
    };
    applyBounCondPeriCPU(f_host[0]);
};

// -----------------------------------------------------------------------
void Field::initFieldSin(double sin_amplitude=1, int sin_period=1, double sin_phase=0) {
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    double r2m=0.25*Nx*Nx+0.25*Ny*Ny;    
    for (int j=0; j<Ny; j++) {
        for (int i=0; i<Nx; i++) {
            int idx=(j+Nby)*(Nx+2*Nbx)+i+Nbx;
            double r2=(i-0.5*Nx)*(i-0.5*Nx)+(j-0.5*Ny)*(j-0.5*Ny);
            f_host[0][idx]=sin_amplitude*sin(2*M_PI*sin_period*i/Nx)*sin(2*M_PI*sin_period*j/Ny);
        };
    };
    applyBounCondPeriCPU(f_host[0]);
};


// ----------------------------------------------------------------------
void Field::setFieldConstCPU(double* f_t, double f_val, int Nx, int Ny, int Nbx, int Nby) {
    setFieldConstCPUCore(f_t, f_val, Nx, Ny, Nbx, Nby);
};

// ----------------------------------------------------------------------
void Field::setFieldConstGPU(double* f_t, double f_val, int Nx, int Ny, int Nbx, int Nby) {
    setFieldConstGPUCore<<<Ny,Nx>>>(f_t, f_val, Nx, Ny, Nbx, Nby);
};


// ----------------------------------------------------------------------
void Field::setRhsTerms(vector<rhsTerm> rhs_terms_t) {
    rhs_terms=rhs_terms_t;
};

// =======================================================================
// Differential operators
// -----------------------------------------------------------------------
// To call this, use:
// FDMCentralO4Iso2D FDM_test;
// phia.getFFuncCPU<FDMCentralO4Iso2D>(phia.laplace, 0, FDM_test, &FDMCentralO4Iso2D::laplace, "new");
template<class FDM_class>
double* Field::getFFuncCPU(double* f_func_ptr, int i_field, FDM_class& FDM_scheme, double (FDM_class::*f_func)(double*,int,int,int,double,double), string method="new") {
    int get_new=1;
    if (method=="old" && f_func_ptr != NULL) {
        get_new=0;
    };

    if (get_new==1) {
        allocField(f_func_ptr, "cpu");
        int Nx=gridNumber().x;
        int Ny=gridNumber().y;
        int Nbx=gridNumberBoun().x;
        int Nby=gridNumberBoun().y;
        int di=1;
        int dj=Nx+2*Nbx;
        double dx=gridSize().x;
        double dy=gridSize().y;

        for (int j=0; j<Ny;j++) {
            for (int i=0; i<Nx; i++) {            
                int idx=(j+Nby)*dj+i+Nbx;
                f_func_ptr[idx]=(FDM_scheme.*f_func)(f[i_field],idx,di,dj,dx,dy);
            };
        };
    };
    return f_func_ptr;
};


// -----------------------------------------------------------------------
// This function is NOT working!!!!
template<class FDM_class>
double* Field::getFFuncGPU(double* f_func_ptr, int i_field, FDM_class& FDM_scheme, double (FDM_class::*f_func)(double*,int,int,int,double,double), string method="new") {
    int get_new=1;
    if (method=="old" && f_func_ptr != NULL) {
        get_new=0;
    };

    if (get_new==1) {
        allocField(f_func_ptr, "gpu");
        int Nx=gridNumber().x;
        int Ny=gridNumber().y;
        int Nbx=gridNumberBoun().x;
        int Nby=gridNumberBoun().y;
        double dx=gridSize().x;
        double dy=gridSize().y;
        
        // getFFuncGPUCore<FDM_class><<<Ny,Nx>>>(f_func_ptr,f[i_field],FDM_scheme,&FDM_class::*f_func,Nx,Ny,Nbx,Nby,dx,dy);
        // getFFuncGPUCore<*FDM_class><<<Ny,Nx>>>(f_func_ptr,f[i_field],FDM_scheme,&f_func,Nx,Ny,Nbx,Nby,dx,dy);
    };

    return f_func_ptr;
};


// -----------------------------------------------------------------------
double* Field::getLaplaceCPU(int i_field, string method="new") {
    int get_new=1;
    if (method=="old" && laplace != NULL) {
        get_new=0;
    };

    if (get_new==1) {
        allocField(laplace, "cpu");
        int Nx=gridNumber().x;
        int Ny=gridNumber().y;
        int Nbx=gridNumberBoun().x;
        int Nby=gridNumberBoun().y;
        int di=1;
        int dj=Nx+2*Nbx;
        double dx=gridSize().x;
        double dy=gridSize().y;

        for (int j=0; j<Ny;j++) {
            for (int i=0; i<Nx; i++) {            
                int idx=(j+Nby)*dj+i+Nbx;
                laplace[idx]=FDM_ptrs[FDM_idx]->laplace(f[i_field],idx,di,dj,dx,dy);
            };
        };
    };
    return laplace;
};

// -----------------------------------------------------------------------
double* Field::getLaplaceGPU(int i_field, string method="new") {
    int get_new=1;
    if (method=="old" && laplace != nullptr) {
        get_new=0;
    };
    
    if (get_new==1) {
        allocField(laplace, "gpu");
        int Nx=gridNumber().x;
        int Ny=gridNumber().y;
        int Nbx=gridNumberBoun().x;
        int Nby=gridNumberBoun().y;
        double dx=gridSize().x;
        double dy=gridSize().y;
        getLaplaceGPUCore<<<Ny,Nx>>>(laplace,f[i_field],FDM_ptrs,FDM_idx,Nx,Ny,Nbx,Nby,dx,dy);
    };
    return laplace;
};


// -----------------------------------------------------------------------
double* Field::getBiLaplaceCPU(int i_field, string method="new") {
    int get_new=1;
    if (method=="old" && bi_laplace != nullptr) {
        get_new=0;
    };

    if (get_new==1) {
        allocField(bi_laplace, "cpu");
        int Nx=gridNumber().x;
        int Ny=gridNumber().y;
        int Nbx=gridNumberBoun().x;
        int Nby=gridNumberBoun().y;
        int di=1;
        int dj=Nx+2*Nbx;
        double dx=gridSize().x;
        double dy=gridSize().y;

        for (int j=0; j<Ny;j++) {
            for (int i=0; i<Nx; i++) {            
                int idx=(j+Nby)*dj+i+Nbx;
                bi_laplace[idx] = FDM_ptrs[FDM_idx]->bi_laplace(f[i_field],idx,di,dj,dx,dy);
            };
        };
    };
    return bi_laplace;
};

// -----------------------------------------------------------------------
double* Field::getBiLaplaceGPU(int i_field, string method="new") {
    int get_new=1;
    if (method=="old" && bi_laplace != nullptr) {
        get_new=0;
    };
    
    if (get_new==1) {
        allocField(bi_laplace, "gpu");
        int Nx=gridNumber().x;
        int Ny=gridNumber().y;
        int Nbx=gridNumberBoun().x;
        int Nby=gridNumberBoun().y;
        double dx=gridSize().x;
        double dy=gridSize().y;
        getBiLaplaceGPUCore<<<Ny,Nx>>>(bi_laplace,f[i_field],FDM_ptrs,FDM_idx,Nx,Ny,Nbx,Nby,dx,dy);
    };
    return bi_laplace;
};


//=======================================================================
void Field::applyBounCondPeriCPU(double* f_t) {
    applyBounCondPeriAnyCPU(f_t);
};

//=======================================================================
void Field::applyBounCondPeriGPU(double* f_t) {
    applyBounCondPeriAnyGPU<<<gridNumber().y,gridNumber().x>>>
        (f_t,
        gridNumber().x,gridNumber().y,
        gridNumberBoun().x,gridNumberBoun().y);
};

//=======================================================================
void Field::applyBounCondPeriAnyCPU(double* f_t) {
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    int dj=Nx+2*Nbx;
    int idx,idx1;
    for (int j=Nby; j<Ny+Nby; j++) {
        for (int i=0; i<Nbx; i++) {
            idx=j*dj+i;
            idx1=idx+Nx;
            f_t[idx]=f_t[idx1];
            f_t[idx1+Nbx]=f_t[idx+Nbx];
        };
    };
    for (int j=0; j<Nby; j++) {
        for (int i=0; i<Nx+2*Nbx; i++) {            
            idx=j*dj+i;
            idx1=idx+dj*Ny;
            f_t[idx]=f_t[idx1];
            f_t[idx1+dj*Nby]=f_t[idx+dj*Nby];
        };
    };

};


// ----------------------------------------------------------------------
void Field::export_conf(string str_t, string device, int include_boun=0) {
    if (device=="cpu") {        
        export_conf_any(f[0],name(),str_t, "cpu", include_boun);
    } else if (device=="gpu") {
        export_conf_any(f[0],name(),str_t, "gpu", include_boun);
    };
}

// ----------------------------------------------------------------------
void Field::export_conf_any(double* f_t, string f_name, string str_t, string location_t="cpu" , int include_boun=0) {
    ofstream conf_file;
    int PrecData=8;
    string conf_file_name="data/"+f_name+"_"+ str_t + ".dat";
    conf_file.open(conf_file_name.c_str() );
    
    int idx;
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    int* idx0=new int [4];

    
    if (location_t=="gpu") {
        allocField(f_temp_host, "cpu");
        hipMemcpy(f_temp_host, f_t, gridNumberAll()*sizeof(double),hipMemcpyDeviceToHost);
    };        
    
    if (include_boun==0) {
        idx0[0]=0;
        idx0[1]=0;
        idx0[2]=0;
        idx0[3]=0;
    } else {
        idx0[0]=-Nbx;
        idx0[1]=Nbx;
        idx0[2]=-Nby;
        idx0[3]=Nby;
    };

    for (int j=idx0[2]; j<Ny+idx0[3]; j++) {
        for (int i=idx0[0]; i<Nx+idx0[1]; i++) {        
            idx=(Nx+2*Nbx)*(j+Nby)+i+Nbx;
            if (location_t=="cpu") {
                conf_file<<setiosflags(ios::scientific) <<setprecision(PrecData) <<f_t[idx]<<endl;
            } else {
                conf_file<<setiosflags(ios::scientific) <<setprecision(PrecData) <<f_temp_host[idx]<<endl;
            };
        }
    }
    conf_file.close();
}

// ----------------------------------------------------------------------
// string Field::equation() {
//     string eqn;
//     if (priority==0) {
//         eqn="p_t "+name()+"=";
//     } else {
//         eqn=name()+"=";
//     };
//     for (auto rhs_term_i : rhsTerms()) {
//         if (rhs_term_i != rhsTerms().begin()) {
            
//         };
//         for (auto f_func_i : rhs_term_i.f_function) {
//             eqn=eqn+
//                 cout<<"*"<<f_func_i.f_operator<<"("<<(*f_func_i.field_ptr).name() <<")";
//                 evalOperator(f_ptr_i,f_func_i,f_func_ptrs[N_funcs],i_field);
//                 N_funcs+=1;
//             };
//             addRHSTerm(f_ptr_i,i_field,rhs_term_i,f_func_ptrs,N_funcs);
//         };

//         if ((*f_ptr_i).priority()>0 && (*f_ptr_i).bounCond()=="periodic") {
//             if (device=="cpu") {
//                 (*f_ptr_i).applyBounCondPeriCPU(i_field);
//             } else if (device=="gpu"){
//                 (*f_ptr_i).applyBounCondPeriGPU(i_field);
//             };
//         };
// };

// -------------------------------------------------------------------
// Copy any field data from CPU to GPU
void Field::updateAnyFieldDev (double* f_dev_ptr, double * f_host_ptr) {

    allocField(f_dev_ptr, "gpu");
    hipMemcpy(f_dev_ptr, f_host_ptr, gridNumberAll()*sizeof(double),hipMemcpyHostToDevice);
};

// -------------------------------------------------------------------
// Copy any field data from GPU to CPU
void Field::updateAnyFieldHost (double* f_host_ptr, double * f_dev_ptr) {    
    hipMemcpy(f_host_ptr, f_dev_ptr, gridNumberAll()*sizeof(double),hipMemcpyDeviceToHost);
};

// -------------------------------------------------------------------
// Copy main field data from CPU to GPU
void Field::updateMainFieldDev () {
    allocField(f[0], "gpu");
    hipMemcpy(f[0], f_host[0], gridNumberAll()*sizeof(double),hipMemcpyHostToDevice);
};

// -------------------------------------------------------------------
// Copy main field data from GPU to CPU
void Field::updateMainFieldHost () {
    allocField(f[0], "cpu");
    hipMemcpy(f_host[0], f[0], gridNumberAll()*sizeof(double),hipMemcpyDeviceToHost);
};

// ------------------------------------------------------------------
void Field::allocField (double* &f_t, string location) {
    if (f_t==NULL) {
        if (location=="cpu") {
            f_t=new double[gridNumberAll()];
        } else if (location=="gpu") {
            hipMalloc(&f_t, gridNumberAll()*sizeof(double));
        };
    };    
};

// =================================================================

#endif
