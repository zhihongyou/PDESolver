#include "hip/hip_runtime.h"
#ifndef FIELDCLASS_INITIALCONDITION_CU
#define FIELDCLASS_INITIALCONDITION_CU

#include <iostream> 
#include <vector>
#include <fstream>
#include <random>
#include <math.h>
#include <cmath>
#include <sstream>
#include <iomanip>

using namespace std;
using std::default_random_engine;
using std::uniform_int_distribution;
using std::uniform_real_distribution;
int seed = time(0);
default_random_engine rng(seed);


// -----------------------------------------------------------------------
void Field::initFieldConst(double f_value) {    
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    uniform_real_distribution<double> randUR;
    for (int j=0; j<Ny; j++) {
        for (int i=0; i<Nx; i++) {
            int idx=(j+Nby)*(Nx+2*Nbx)+i+Nbx;
            f_host[0][idx]=f_value+0.0001*(randUR(rng)-0.5);
        };
    };
    applyBounCondPeriCPU(f_host[0]);
};

// -----------------------------------------------------------------------
void Field::initFieldGaus(double r_center, double r_decay, double gaus_amplitude) {
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    double dx2=gridSize().x*gridSize().x;
    double dy2=gridSize().y*gridSize().y;
    double rd2=r_decay*r_decay;
    uniform_real_distribution<double> randUR;
    for (int j=0; j<Ny; j++) {
        for (int i=0; i<Nx; i++) {
            int idx=(j+Nby)*(Nx+2*Nbx)+i+Nbx;
            double r2=dx2*(i-0.5*Nx)*(i-0.5*Nx)+dy2*(j-0.5*Ny)*(j-0.5*Ny);
            f_host[0][idx]=gaus_amplitude*(exp(-r2/rd2))+0.0001*(randUR(rng)-0.5);
        };
    };
    applyBounCondPeriCPU(f_host[0]);
};

// -----------------------------------------------------------------------
void Field::initFieldSin(double sin_amplitude=1, int sin_period=1, double sin_phase=0) {
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    double r2m=0.25*Nx*Nx+0.25*Ny*Ny;
    uniform_real_distribution<double> randUR;
    for (int j=0; j<Ny; j++) {
        for (int i=0; i<Nx; i++) {
            int idx=(j+Nby)*(Nx+2*Nbx)+i+Nbx;
            double r2=(i-0.5*Nx)*(i-0.5*Nx)+(j-0.5*Ny)*(j-0.5*Ny);
            f_host[0][idx]=sin_amplitude*sin(4*M_PI*sin_period*i/Nx)*sin(2*M_PI*sin_period*j/Ny)+0.0001*(randUR(rng)-0.5);           
        };
    };
    applyBounCondPeriCPU(f_host[0]);
};


// =================================================================

#endif
