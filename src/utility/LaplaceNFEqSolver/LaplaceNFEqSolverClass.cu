#include "hip/hip_runtime.h"
#ifndef LAPLACENFEQSOLVERCLASS_CU
#define LAPLACENFEQSOLVERCLASS_CU

#include <iostream> 
#include <vector>
#include <string>
#include <map>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "LaplaceNFEqSolverClass.h"


using namespace std;


// =============================================================
// Constructors
LaplaceNFEqSolver::LaplaceNFEqSolver () {};


// ------------------------------------------------------------
LaplaceNFEqSolver::LaplaceNFEqSolver (Mesh* mesh_ptr_t) {
    // cout <<"haha"<<endl;
    initLaplaceNFEqSolver(mesh_ptr_t);
    // cout <<"haha"<<endl;
};


// ------------------------------------------------------------
LaplaceNFEqSolver::LaplaceNFEqSolver (Mesh* mesh_ptr_t, string name_t) {
    name=name_t;
    initLaplaceNFEqSolver(mesh_ptr_t);
};


//===============================================================
void LaplaceNFEqSolver::initLaplaceNFEqSolver(Mesh* mesh_ptr_t) {
    int Nx=(*mesh_ptr_t).host.grid_number.x;    
    int Ny=(*mesh_ptr_t).host.grid_number.y;
    if (k2s_host==NULL) {
        k2s_host=new double[Nx*Ny];
        hipMalloc((void **)&k2s_dev, (Nx*Ny)*sizeof(double));
        hipMalloc((void **)&phi_complex, sizeof(hipfftDoubleComplex)*Nx*Ny);
        hipfftPlan2d(&cufftPlan, Ny, Nx, HIPFFT_Z2Z);
    }
    mesh_ptr=mesh_ptr_t;
    double prefactors_t[2]={0, 1};
    setLaplaceNFEqSolver(1, prefactors_t);
}


// ============================================================
void LaplaceNFEqSolver::setLaplaceNFEqSolver(int max_power_t, double* prefactors_t) {
    max_power=max_power_t;
    for (int i=0; i<=max_power; i++) {
        prefactors[i]=prefactors_t[i];
    }
    setk2s();
};


//==============================================================
void LaplaceNFEqSolver::setk2s() {
    double kx,ky;
    int Nx=(*mesh_ptr).host.grid_number.x;
    int Ny=(*mesh_ptr).host.grid_number.y;
    double dx=(*mesh_ptr).host.grid_size.x;
    double dy=(*mesh_ptr).host.grid_size.y;
    
    for (int i=0; i<Ny; i++){
        ky = 2*Pi*i/(Ny*dy+0.0);
        if (i>=Ny/2) {
            ky=2*Pi*(i-Ny)/(Ny*dy+0.0);
        }
        for (int j=0; j<Nx; j++){
            kx=2*Pi*j/(Nx*dx+0.0);
            if (j>=Nx/2) {
                kx=2*Pi*(j-Nx)/(Nx*dx+0.0);
            }
            int idx=i*Nx+j;
	    k2s_host[idx]=prefactors[0];
	    for (int k=1; k<=max_power; k++) {
	      k2s_host[idx]=k2s_host[idx]+prefactors[k]*pow(-kx*kx-ky*ky, k);
	    }
        }
    }
    if (abs(prefactors[0])<0.0000000000000001) {
      k2s_host[0]=1;
    } else {
      k2s_host[0]=prefactors[0];
    };
    hipMemcpy(k2s_dev,k2s_host,sizeof(double)*Nx*Ny,hipMemcpyHostToDevice);
};


// --------------------------------------------------------------
void LaplaceNFEqSolver::solveLaplaceNFEq (double* phi, double* f) {
    int Nx=(*mesh_ptr).host.grid_number.x;
    int Ny=(*mesh_ptr).host.grid_number.y;
    int Nbx=(*mesh_ptr).host.grid_number_boun.x;
    int Nby=(*mesh_ptr).host.grid_number_boun.y;

    // Assign f to the real part of phi_complex
    solveLaplaceNFEqCoreGPU<<<Ny,Nx>>>(phi_complex, phi, f, k2s_dev, Nx, Ny, Nbx, Nby, 0);
    // Fourier transform phi_complex
    hipfftExecZ2Z(cufftPlan,phi_complex,phi_complex,HIPFFT_FORWARD);
    // Set phi_complex=phi_complex/k2s
    solveLaplaceNFEqCoreGPU<<<Ny,Nx>>>(phi_complex, phi, f, k2s_dev, Nx, Ny, Nbx, Nby, 1);
    // Inverse Fourier transform phi_complex
    hipfftExecZ2Z(cufftPlan,phi_complex,phi_complex,HIPFFT_BACKWARD);
    // Assign the real part of phi_complex to phi
    solveLaplaceNFEqCoreGPU<<<Ny,Nx>>>(phi_complex, phi, f, k2s_dev, Nx, Ny, Nbx, Nby, 2);
    // applyBounCondPeriGPU(phi);    
};


// ==============================================================

#endif
