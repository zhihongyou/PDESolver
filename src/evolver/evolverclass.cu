#include "hip/hip_runtime.h"
#ifndef EVOLVERCLASS_CPP
#define EVOLVERCLASS_CPP

#include <iostream> 
#include <vector>
#include "evolverclass.h"
#include "evolverclassGPU.cu"

using namespace std;


// ======================================================================

// ----------------------------------------------------------------------
// Update system over time. This is the global control.
void Evolver::run() {
    cout <<"Start running simulation ..." <<endl;
    time_start_wall = clock();
    
    // Initiate function calls and pointers.
    initEvolver();
    
    // Export initial configuration.
    for (auto f_ptr_i : (*system_ptr).field_ptrs ) {
        if ((*f_ptr_i).expoData() =="on") {
            (*f_ptr_i).export_conf("0",device,1);
        };
    };
    // cout <<"Export 0 successfully"<<endl;
    

    // Running through time
    for (time_now=time_start; time_now<=time_stop+0.1*time_step; time_now+=time_step) {

        // Move a step forward.
        if (scheme=="EulerForward") {
            EulerForward();
        };

        // Reach export time.
        if ((int) (time_now/time_export) > (int) ((time_now-time_step)/time_export)) {
            // Export field configuration.
            int te=floor(time_now/time_export);    
            string str_t=to_string(te);
            for (auto f_ptr_i : (*system_ptr).field_ptrs ) {
                if ((*f_ptr_i).expoData() =="on") {
                    (*f_ptr_i).export_conf(str_t,device,1);
                    (*f_ptr_i).export_conf_any((*f_ptr_i).rhs[0],"rhs",str_t,device,1);
                    (*f_ptr_i).export_conf_any((*f_ptr_i).laplace,"laplace",str_t,device,1);                    
                };        
            };
            // Show progress of simulation.
            showProgress();
        };
        
    };
    // Simulation finished.
    time_stop_wall = clock();
    std::cout.flush();
    cout <<"Simulation finished. Total spent:";
    cout<<(time_stop_wall-time_start_wall)/CLOCKS_PER_SEC<<" seconds.";
    cout <<"                                                        ";
    cout<<endl;
};

// ----------------------------------------------------------------------
void Evolver::initEvolver() {
    initFields();
    initRHSs();
};


// ----------------------------------------------------------------------
void Evolver::initRHSs() {
    
    // Loop over fields.    
    for (auto f_ptr_i : (*system_ptr).field_ptrs ) {
        int num_grid=(*f_ptr_i).gridNumberAll();
        int num_terms=(*f_ptr_i).rhsTerms().size();        
        int num_funcs=0;
        for (auto rhs_term_i : (*f_ptr_i).rhsTerms()) {
            num_funcs+=rhs_term_i.f_funcs.size();
        };
        (*f_ptr_i).rhs_ptrs_host.num_terms=new int[2];
        (*f_ptr_i).rhs_ptrs_host.num_funcs_1term=new int[num_terms];
        (*f_ptr_i).rhs_ptrs_host.prefactors=new double[num_terms];
        (*f_ptr_i).rhs_ptrs_host.schemes=new int[num_funcs];
        (*f_ptr_i).rhs_ptrs_host.f_func_ptrs=new double * [num_funcs];
        
        
        // Loop over terms on the RHS of each field.
        // cout <<"For field "<<(*f_ptr_i).name()<<": "<<endl;
        int i_func=0;
        int i_term=0;
        int num_terms_expl=0;        
        // Add explicit terms
        for (auto rhs_term_i : (*f_ptr_i).rhsTerms()) {            
            // double* f_func_ptrs[10];
            int i_func_1term=0;
            if (rhs_term_i.scheme=="explicit") {
                num_terms_expl+=1;
                // (*f_ptr_i).rhs_ptrs_host.schemes[i_term]=1;
            
                // Evaluate each operator applied on field
                for (auto f_func_i : rhs_term_i.f_funcs) {
                    if (f_func_i.f_operator=="1") {                
                        (*f_ptr_i).rhs_ptrs_host.f_func_ptrs[i_func]=(*f_func_i.field_ptr).f_now;
                    };
                    if (f_func_i.f_operator=="laplace") {
                        (*f_ptr_i).rhs_ptrs_host.f_func_ptrs[i_func]=(*f_func_i.field_ptr).laplace;
                    };
                    if (f_func_i.f_operator=="bi_laplace") {
                        (*f_ptr_i).rhs_ptrs_host.f_func_ptrs[i_func]=(*f_func_i.field_ptr).bi_laplace;
                    };
                
                    // Add this term to function list of this field. Each function appears only once.
                    int toAdd=1;
                    // Check if function is already in the list
                    for (auto f_func_i1 : (*f_ptr_i).f_funcs_rhs) {
                        if (f_func_i.f_operator == f_func_i1.f_operator) {
                            toAdd=0;
                        };
                    };
                    if (toAdd==1) {
                        (*f_ptr_i).f_funcs_rhs.push_back(f_func_i);
                    };

                    i_func+=1;
                    i_func_1term+=1;
                };
                (*f_ptr_i).rhs_ptrs_host.num_funcs_1term[i_term]=i_func_1term;
                (*f_ptr_i).rhs_ptrs_host.prefactors[i_term]=rhs_term_i.prefactor;
                i_term+=1;
            };
        };

        int num_terms_impl=0;
        for (auto rhs_term_i : (*f_ptr_i).rhsTerms()) {            
            if (rhs_term_i.scheme=="semiImplicit") {
                int i_func_1term=0;
                num_terms_impl+=1;
                // (*f_ptr_i).rhs_ptrs_host.schemes[i_term]=-1;                
                // Evaluate each operator applied on field
                for (auto f_func_i : rhs_term_i.f_funcs) {                             
                    if (f_func_i.f_operator=="1") {
                        (*f_ptr_i).rhs_ptrs_host.f_func_ptrs[i_func]=(*f_func_i.field_ptr).f_now;
                    };
                    if (f_func_i.f_operator=="laplace") {
                        (*f_ptr_i).rhs_ptrs_host.f_func_ptrs[i_func]=(*f_func_i.field_ptr).laplace;
                    };
                    if (f_func_i.f_operator=="bi_laplace") {
                        (*f_ptr_i).rhs_ptrs_host.f_func_ptrs[i_func]=(*f_func_i.field_ptr).bi_laplace;
                    };
                
                    // Add this term to function list of this field. Each function appears only once.                    
                    int toAdd=1;
                    // Check if function is already in the list
                    for (auto f_func_i1 : (*f_ptr_i).f_funcs_rhs) {
                        if (f_func_i.f_operator == f_func_i1.f_operator) {
                            toAdd=0;
                        };
                    };
                    if (toAdd==1) {                            
                        (*f_ptr_i).f_funcs_rhs.push_back(f_func_i);
                    };

                    i_func+=1;
                    i_func_1term+=1;
                };
                (*f_ptr_i).rhs_ptrs_host.num_funcs_1term[i_term]=i_func_1term;
                (*f_ptr_i).rhs_ptrs_host.prefactors[i_term]=rhs_term_i.prefactor;
                i_term+=1;
            };
        };
        
        (*f_ptr_i).rhs_ptrs_host.num_terms[0]=num_terms_expl;
        (*f_ptr_i).rhs_ptrs_host.num_terms[1]=num_terms_impl;        

        if (device=="gpu") {
            
            // Copy these values to device.
            hipMalloc(&(*f_ptr_i).rhs_ptrs_dev.num_terms,2*sizeof(int));
            hipMalloc(&(*f_ptr_i).rhs_ptrs_dev.num_funcs_1term,num_terms*sizeof(int));
            hipMalloc(&(*f_ptr_i).rhs_ptrs_dev.schemes,num_terms*sizeof(int));
            hipMalloc(&(*f_ptr_i).rhs_ptrs_dev.prefactors,num_terms*sizeof(double));
            hipMalloc(&(*f_ptr_i).rhs_ptrs_dev.f_func_ptrs,num_funcs*sizeof(double*));
            hipMemcpy((*f_ptr_i).rhs_ptrs_dev.num_terms, (*f_ptr_i).rhs_ptrs_host.num_terms, 2*sizeof(int),hipMemcpyHostToDevice);
            hipMemcpy((*f_ptr_i).rhs_ptrs_dev.num_funcs_1term, (*f_ptr_i).rhs_ptrs_host.num_funcs_1term, num_terms*sizeof(int),hipMemcpyHostToDevice);
            hipMemcpy((*f_ptr_i).rhs_ptrs_dev.schemes, (*f_ptr_i).rhs_ptrs_host.schemes, num_terms*sizeof(int),hipMemcpyHostToDevice);
            hipMemcpy((*f_ptr_i).rhs_ptrs_dev.prefactors, (*f_ptr_i).rhs_ptrs_host.prefactors, num_terms*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy((*f_ptr_i).rhs_ptrs_dev.f_func_ptrs, (*f_ptr_i).rhs_ptrs_host.f_func_ptrs, num_funcs*sizeof(double*),hipMemcpyHostToDevice);
        };
    };    
};

// ----------------------------------------------------------------------
// Allocate memory for fields and field functions.
// Location of memory depends on device of the evolver.
void Evolver::initFields () {    

    // Initiate fields.
    for (auto f_ptr_i : (*system_ptr).field_ptrs ) {
        int num_grid=(*f_ptr_i).gridNumberAll();
        if (device=="cpu") {            
            for (int i_f_copy=0; i_f_copy<num_field_copy; i_f_copy++) {
                (*f_ptr_i).f[i_f_copy]=new double[num_grid];
            };
            for (int idx=0; idx<num_grid; idx++) {
                (*f_ptr_i).f[0][idx]=(*f_ptr_i).f_host[0][idx];
            };
            
        } else if (device=="gpu") {
            (*f_ptr_i).f_temp_host=new double[(*f_ptr_i).gridNumberAll()];
            for (int i_f_copy=0; i_f_copy<num_field_copy; i_f_copy++) {
                hipMalloc(&(*f_ptr_i).f[i_f_copy], num_grid*sizeof(double));
            };
            hipMemcpy((*f_ptr_i).f[0], (*f_ptr_i).f_host[0], num_grid*sizeof(double),hipMemcpyHostToDevice);            
        };
    };

    // Initiate field functions.
    for (auto f_ptr_i : (*system_ptr).field_ptrs ) {
        int num_grid=(*f_ptr_i).gridNumberAll();
        for (auto rhs_term_i : (*f_ptr_i).rhsTerms()) {            
            for (auto f_func_i : rhs_term_i.f_funcs) {                
                if (f_func_i.f_operator=="1") {
                    if ((*f_func_i.field_ptr).f_now==NULL) {
                        if (device=="cpu") {
                            (*f_func_i.field_ptr).f_now=new double[num_grid];
                        } else {
                            hipMalloc(&(*f_func_i.field_ptr).f_now, num_grid*sizeof(double));
                        };
                    };
                };
                if (f_func_i.f_operator=="laplace") {                                        
                    if ((*f_func_i.field_ptr).laplace==NULL) {
                        if (device=="cpu") {
                            (*f_func_i.field_ptr).laplace=new double[num_grid];
                        } else {
                            hipMalloc(&(*f_func_i.field_ptr).laplace, num_grid*sizeof(double));
                        };
                    };
                };

                if (f_func_i.f_operator=="bi_laplace") {
                    if ((*f_func_i.field_ptr).bi_laplace==NULL) {
                        if (device=="cpu") {
                            (*f_func_i.field_ptr).bi_laplace=new double[num_grid];
                        } else {
                            hipMalloc(&(*f_func_i.field_ptr).bi_laplace, num_grid*sizeof(double));
                        };
                    };
                };
            };
        };
    };                
    
};

// ----------------------------------------------------------------------
// Euler forward scheme to evolve over time.
void Evolver::EulerForward() {
    
    for (auto f_ptr_i : (*system_ptr).field_ptrs ) {        
        getRHS(f_ptr_i,0);
    };
                
    fieldsUpdate(0,0,0);
};

// ----------------------------------------------------------------------
void Evolver::getRHS(Field* f_ptr_i, int i_field) {

    allocateRHS(f_ptr_i,i_field);

    evalFieldFuncs(f_ptr_i,i_field);    

    // (*f_ptr_i).export_conf_any((*f_ptr_i).f[0],"phi","0",device,1);
    // (*f_ptr_i).export_conf_any((*f_ptr_i).laplace,"laplace","0",device,1);    
    // (*f_ptr_i).export_conf_any((*f_ptr_i).rhs[0],"rhs", "0",device,1);
    
    updateRHS(f_ptr_i,i_field);

    // (*f_ptr_i).export_conf_any((*f_ptr_i).f[0],"phi","3",device,1);
    // (*f_ptr_i).export_conf_any((*f_ptr_i).laplace,"laplace","3",device,1);    
    // (*f_ptr_i).export_conf_any((*f_ptr_i).rhs[0],"rhs", "3",device,1);        
    // cout <<"RHS6"<<endl;
    
    if ((*f_ptr_i).priority()>0 && (*f_ptr_i).bounCond()=="periodic") {
        if (device=="cpu") {
            (*f_ptr_i).applyBounCondPeriCPU((*f_ptr_i).f[i_field]);
        } else if (device=="gpu"){
            (*f_ptr_i).applyBounCondPeriGPU((*f_ptr_i).f[i_field]);
        };
    };        

};

// ----------------------------------------------------------------------
void Evolver::allocateRHS(Field* f_ptr_t, int i_field) {
    int Nx=(*f_ptr_t).gridNumber().x;
    int Ny=(*f_ptr_t).gridNumber().y;
    int Nbx=(*f_ptr_t).gridNumberBoun().x;
    int Nby=(*f_ptr_t).gridNumberBoun().y;
    if ((*f_ptr_t).priority()==0) {
        if (device=="cpu") {
            if ((*f_ptr_t).rhs[i_field] == NULL) {
                (*f_ptr_t).rhs[i_field]=new double[(*f_ptr_t).gridNumberAll()];
            };
            if ((*f_ptr_t).lhs[i_field] == NULL) {
                (*f_ptr_t).lhs[i_field]=new double[(*f_ptr_t).gridNumberAll()];
            };
        } else if (device=="gpu") {
            if ((*f_ptr_t).rhs[i_field] == NULL) {
                hipMalloc(&(*f_ptr_t).rhs[i_field],
                (*f_ptr_t).gridNumberAll()*sizeof(double));
            };
            if ((*f_ptr_t).lhs[i_field] == NULL) {
                hipMalloc(&(*f_ptr_t).lhs[i_field],
                (*f_ptr_t).gridNumberAll()*sizeof(double));
            };
        };
        
    } else {
        if ((*f_ptr_t).f[i_field] == NULL) {
            if (device=="cpu") {
                (*f_ptr_t).f[i_field]=new double[(*f_ptr_t).gridNumberAll()];
            } else if (device=="gpu") {            
                hipMalloc(&(*f_ptr_t).f[i_field],
                (*f_ptr_t).gridNumberAll()*sizeof(double));
            };
        };
    };
};


// -----------------------------------------------------------------------
void Evolver::evalFieldFuncs(Field* f_ptr_i, int i_field) {

    for (auto f_func_i : (*f_ptr_i).f_funcs_rhs ) {        
        
        if (f_func_i.f_operator == "1") {
            (*f_func_i.field_ptr).f_now=(*f_func_i.field_ptr).f[i_field];
        };
        
        if (f_func_i.f_operator == "laplace") {
            if (device=="cpu") {
                (*f_func_i.field_ptr).getLaplaceCPU(i_field,"new");
            } else if (device=="gpu") {                
                (*f_func_i.field_ptr).getLaplaceGPU(i_field,"new");
            };
        };
        
        if (f_func_i.f_operator == "bi_laplace") {
            if (device=="cpu") {
                (*f_func_i.field_ptr).getLaplaceCPU(i_field,"new");
            } else if (device=="gpu") {
                (*f_func_i.field_ptr).getLaplaceGPU(i_field,"new");
            };
        };
    };
        
};


// -----------------------------------------------------------------------
void Evolver::updateRHS(Field* f_ptr_t, int i_field) {
    int Nx=(*f_ptr_t).gridNumber().x;
    int Ny=(*f_ptr_t).gridNumber().y;
    int Nbx=(*f_ptr_t).gridNumberBoun().x;
    int Nby=(*f_ptr_t).gridNumberBoun().y;    
    double* rhs_temp;
    double* lhs_temp;
    
    if ((*f_ptr_t).priority() == 0) {
        rhs_temp=(*f_ptr_t).rhs[i_field];
        lhs_temp=(*f_ptr_t).lhs[i_field];
    } else {
        if (device == "cpu") {
            rhs_temp=(*f_ptr_t).f[i_field];
            lhs_temp=(*f_ptr_t).f[i_field];
        } else if (device == "gpu") {
            rhs_temp=(*f_ptr_t).f[i_field];
            lhs_temp=(*f_ptr_t).f[i_field];
        };
    };    
    
    if (rhs_temp == NULL) {
        if (device=="cpu") {
            rhs_temp=new double[(*f_ptr_t).gridNumberAll()];
        } else if (device=="gpu") {            
            hipMalloc(&rhs_temp,
            (*f_ptr_t).gridNumberAll()*sizeof(double));
        };
    };

    if (lhs_temp == NULL) {
        if (device=="cpu") {
            lhs_temp=new double[(*f_ptr_t).gridNumberAll()];
        } else if (device=="gpu") {
            hipMalloc(&lhs_temp,
            (*f_ptr_t).gridNumberAll()*sizeof(double));
        };
    };

    // (*f_ptr_t).export_conf_any((*f_ptr_t).laplace,"laplace","1",device,1);
    // (*f_ptr_t).export_conf_any((*f_ptr_t).f[0],"phi","1",device,1);
    // (*f_ptr_t).export_conf_any((*f_ptr_t).rhs[0],"rhs", "1",device,1);
    
    if (device=="cpu") {
        updateRHSCoreCPU((*f_ptr_t).rhs_ptrs_host, rhs_temp, lhs_temp, Nx, Ny, Nbx, Nby);
    } else if (device=="gpu") {        
        updateRHSCoreGPU<<<Ny,Nx>>>((*f_ptr_t).rhs_ptrs_dev, rhs_temp, lhs_temp, Nx, Ny, Nbx, Nby);
    };

    // (*f_ptr_t).export_conf_any((*f_ptr_t).laplace,"laplace","2",device,1);
    // (*f_ptr_t).export_conf_any((*f_ptr_t).f[0],"phi","2",device,1);
    // (*f_ptr_t).export_conf_any((*f_ptr_t).rhs[0],"rhs", "2",device,1);    
    
};

// ----------------------------------------------------------------------
void Evolver::updateRHSCoreCPU(rhsPtrs rhs_ptrs, double* rhs_temp, double* lhs_temp, int Nx, int Ny, int Nbx, int Nby) {
    
    double temp;
    for (int j=0; j<Ny;j++) {
        for (int i=0; i<Nx; i++) {
            int idx=(j+Nby)*(Nx+2*Nbx)+i+Nbx;
            rhs_temp[idx]=0;
            lhs_temp[idx]=0;
            int i_func=0;
            for (int i_term=0; i_term<rhs_ptrs.num_terms[0]; i_term++) {
                temp=rhs_ptrs.prefactors[i_term];
                for (int i_func1=0; i_func1<rhs_ptrs.num_funcs_1term[i_term]; i_func1++) {
                    temp=temp*rhs_ptrs.f_func_ptrs[i_func][idx];
                    i_func+=1;
                };
                rhs_temp[idx]+=temp;                
            };
            for (int i_term=rhs_ptrs.num_terms[0]; i_term<rhs_ptrs.num_terms[0]+rhs_ptrs.num_terms[1]; i_term++) {
                double temp=rhs_ptrs.prefactors[i_term];
                for (int i_func1=0; i_func1<rhs_ptrs.num_funcs_1term[i_term]-1; i_func1++) {
                    temp=temp*rhs_ptrs.f_func_ptrs[i_func][idx];
                    i_func+=1;
                };
                lhs_temp[idx]+=temp;
            };
        };
    };    
};


// ----------------------------------------------------------------------
void Evolver::fieldsUpdate(int i_f_new, int i_f_old, int i_df) {
    for (auto f_ptr_i : (*system_ptr).field_ptrs ) {
        if ((*f_ptr_i).priority() ==0) {
            if (device=="cpu") {
                fieldUpdateCPU(f_ptr_i,i_f_new,i_f_old,i_df,time_step);                
                if ((*f_ptr_i).bounCond()=="periodic") {
                    (*f_ptr_i).applyBounCondPeriCPU((*f_ptr_i).f[i_f_new]);
                };
            } else if (device=="gpu") {
                fieldUpdateGPU(f_ptr_i,i_f_new,i_f_old,i_df,time_step);
                if ((*f_ptr_i).bounCond()=="periodic") {
                    (*f_ptr_i).applyBounCondPeriGPU((*f_ptr_i).f[i_f_new]);
                };
                
            };            
        };        
    };
};

// ----------------------------------------------------------------------
void Evolver::fieldUpdateCPU(Field* f_ptr_t, int i_f_new, int i_f_old, int i_df, double time_step_t) {
    int Nx=(*f_ptr_t).gridNumber().x;
    int Ny=(*f_ptr_t).gridNumber().y;
    int Nbx=(*f_ptr_t).gridNumberBoun().x;
    int Nby=(*f_ptr_t).gridNumberBoun().y;
    
    for (int j=0; j<Ny;j++) {
        for (int i=0; i<Nx; i++) {        
            int idx=(j+Nby)*(Nx+2*Nbx)+i+Nbx;
            (*f_ptr_t).f[i_f_new][idx]=((*f_ptr_t).f[i_f_old][idx]+(*f_ptr_t).rhs[i_df][idx]*time_step_t)/(1+(*f_ptr_t).lhs[i_df][idx]*time_step_t);
        };
    };
};

// ----------------------------------------------------------------------
void Evolver::fieldUpdateGPU(Field* f_ptr_t, int i_f_new, int i_f_old, int i_df, double time_step_t) {
    
    int Nx=(*f_ptr_t).gridNumber().x;
    int Ny=(*f_ptr_t).gridNumber().y;
    int Nbx=(*f_ptr_t).gridNumberBoun().x;
    int Nby=(*f_ptr_t).gridNumberBoun().y;
    fieldUpdateGPUCore<<<Ny,Nx>>>((*f_ptr_t).f[i_f_new], (*f_ptr_t).f[i_f_old], (*f_ptr_t).rhs[i_df], (*f_ptr_t).lhs[i_df], time_step_t, Nx, Ny, Nbx, Nby);
};


// ----------------------------------------------------------------------
void Evolver::showProgress() {
  // Print progress.
    double progress=(time_now-time_start)/(time_stop-time_start);
    int barWidth = 50;
    time_now_wall = clock();
    double time_used_wall=double(time_now_wall-time_start_wall)/CLOCKS_PER_SEC;
  
    std::cout << "Progress: ";  
    std::cout << "[";
    int pos = barWidth * progress;
    for (int i = 0; i < barWidth; ++i) {
        if (i < pos) std::cout << "=";
        else if (i == pos) std::cout << ">";
        else std::cout << " ";
    }
    std::cout << "] " << int(progress * 100.0) << " %";
    if (time_now==0) {
        std::cout <<"\r";
    } else {
        std::cout << ".  " << floor(time_used_wall/progress*(1-progress)) << "s remains.\r";
    }

    std::cout.flush();
}


// ======================================================================

#endif
