#include "hip/hip_runtime.h"
#include <iostream> 
#include <vector>
#include <list>
#include <cstdio>
#include <string>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime_api.h"
#include <cmath>
#include <ctime>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
// #include "src/mesh/meshclass.cuh"
// #include "src/field/fieldclass.cu"
// #include "src/system/systemclass.cpp"
// #include "src/evolver/evolverclass.h"
#include "src/evolver/evolverclass.cu"
// #include "src/utility/finiteDifference.h"
// #include "src/utility/finiteDifferenceCentralO2Isotropic.h"

using namespace std;

__global__ void testGPU (FiniteDifference** FDM_ptrs, int *FDM_idx) {
    FDM_idx[0]=FDM_idx[0]+1;
};


// ======================================================================
int main() {
    // Generating a new system.
    System mySys;
    // Generating a new mesh.
    Mesh mesh(2);
    // Add mesh to system.
    mySys.mesh_ptr=&mesh;
    
    // Creating fields.
    Field mua(&mesh, "mua", 0, 1, "periodic", "Gaussian", "on");
    Field phia(&mesh, "phia", 0, 0, "periodic", "sin", "on");
    Field mub(&mesh, "mub", 0, 1, "periodic", "Gaussian", "on");
    Field phib(&mesh, "phib", 0, 0, "periodic", "sin", "on");

    // Set field equations.
    mua.setRhsTerms({
        {-1,{{"laplace",&phia}},"explicit"},
        {-0.2,{{"1",&phia}},"explicit"},
        {1,{{"1",&phia},{"1",&phia},{"1",&phia}},"explicit"}
    });

    mub.setRhsTerms({
        {-1,{{"laplace",&phib}},"explicit"},
        {-0.2,{{"1",&phib}},"explicit"},
        {1,{{"1",&phib},{"1",&phib},{"1",&phib}},"explicit"}
    });
    
    phia.setRhsTerms({
        {1,{{"laplace",&mua}},"explicit"},
        {-0.1,{{"laplace",&phib}},"explicit"}
    });

    phib.setRhsTerms({
        {1,{{"laplace",&mub}},"explicit"},
        {0.1,{{"laplace",&phia}},"explicit"}
    });
    
    // Add fields to the system.
    mySys.field_ptrs.push_back(&mua);
    mySys.field_ptrs.push_back(&mub);
    mySys.field_ptrs.push_back(&phia);
    mySys.field_ptrs.push_back(&phib);
    // Print system information.
    // mySys.printSysInfo();
    
    // Creating an evolver:
    string device="gpu";
    string FDScheme="CentralDifferenceO4Iso2D";
    Evolver evolver(&mySys,0,10000,0.02,100,device,"EulerForward",FDScheme);
    evolver.run();

    // -----------------------------------------------------------
    // evolver.initEvolver();
    // FDMCentralO4Iso2D FDM_test;
    // cout<<"export 1"<<endl;
    // phia.export_conf_any(phia.f[0],"phia","1", device, 1);
    // phia.export_conf_any(phia.laplace,"phia_laplace","1", device, 1);
    // phia.export_conf_any(phia.one_over_f,"phia_laplace","1", device, 1);
    // cout<<"Get Laplace."<<endl;
    
    // for (int i=0; i<100000;i++) {
        // phia.getLaplaceCPU(0,"new");
        // phia.getFFuncCPU<FDMCentralO4Iso2D>(phia.laplace, 0, FDM_test, &FDMCentralO4Iso2D::laplace, "new");
    // };
    // phia.getFFuncGPU<double>(phia.one_over_f, 0, FieldFunction::oneOverF, "new");
    // cout<<"Export 2"<<endl;
    // phia.export_conf_any(phia.f[0],"phia","2", device, 1);
    // cout<<"Export 1/f"<<endl;
    // if (phia.one_over_f==NULL) {
    //     cout<<"1/f is NULL"<<endl;
    // };
    // phia.export_conf_any(phia.laplace,"phia_laplace","2", device, 1);
    // phia.export_conf_any(phia.one_over_f,"phia_laplace","2", device, 1);
    // // evolver.evalFieldFuncs(&mua,0);
    // phia.getLaplaceGPU(0,"new");
    // evolver.updateRHS(&mua,0);
    
    // mua.applyBounCondPeriGPU(mua.f[0]);
    // evolver.evalFieldFuncs(&phia,0);
    // evolver.updateRHS(&phia,0);
    // phia.export_conf_any(phia.f[0],"phia","2", device, 1);
    // phia.export_conf_any(phia.laplace,"phia_laplace","2", device, 1);
    // phia.export_conf_any(phia.rhs[0],"phia_rhs","2", device, 1);
    // mua.export_conf_any(mua.f[0],"mua","2", device, 1);
    // mua.export_conf_any(mua.laplace,"mua_laplace","2", device, 1);

    // -----------------------------------------------------------
    
    return 0;
};

