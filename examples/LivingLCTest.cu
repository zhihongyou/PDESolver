#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream> 
#include <vector>
#include <list>
#include <cstdio>
#include <string>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime_api.h"
#include <cmath>
#include <ctime>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "userDefinedFunction.h"
#include "../src/evolver/evolverclass.cu"

using namespace std;


// ======================================================================
int main() {

    // Simulation parameters
    string direExpo="data/";
    string device="gpu";
    string FDMScheme="CentralDifferenceO4Iso2D";
    string timeScheme="RK4";
    double dt=0.001;
    double T=10;
    double dtExpo=1;
    int    NGrid=128;
    double L=NGrid*1.0;       
    
    // Generating a new system.
    System mySys(direExpo);
    // Generating a new mesh.
    Mesh mesh(2);
    mesh.setGridNumber(NGrid,NGrid,1);
    mesh.setBoxSize(L,L,1);
    // Add mesh to system.
    mySys.mesh_ptr=&mesh;
    
    // Creating fields.
    LivingLC livingLC(&mesh, "livingLC", 0);    
    Field phi(&mesh, "phi",1);
    livingLC.Pxx.initFieldSin(0.5, 2, 0);
    livingLC.Pxy.initFieldSin(0.3, 4, 0);
    
    livingLC.Pxx.setRhsTerms({
        {-1,{{&livingLC.Pxx}}}
    });

    livingLC.Pxy.setRhsTerms({
        {-1,{{&livingLC.Pxy}}}
    });
    
    phi.setRhsTerms({
        {{{"sin",&livingLC.Pxx,{2,0}}}}
    });
    
    // phi.initFieldGaus(L/2, 0.1*L, 1.5);
    mySys.addLivingLC(&livingLC);
    mySys.addField(&phi);
    
    // Print system information.
    mySys.printSysInfo();
    
    // Creating an evolver:    
    Evolver evolver(&mySys,0,T,dt,dtExpo,device,timeScheme,FDMScheme);
    // evolver.initEvolver();
    // Running simulations
    evolver.run();

    // f.export_conf_any(f.rhs[0], "f_rhs", "", device);
    // phi.export_conf_any(phi.rhs[0], "phi_rhs", "", device);
    // psi.export_conf_any(psi.rhs[0], "psi_rhs", "", device);
    
    // -----------------------------------------------------------
    
    return 0;
};

