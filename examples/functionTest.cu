#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream> 
#include <vector>
#include <list>
#include <cstdio>
#include <string>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime_api.h"
#include <cmath>
#include <ctime>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "userDefinedFunction.h"
#include "../src/evolver/evolverclass.cu"

using namespace std;


// ======================================================================
int main() {

    // Simulation parameters
    string direExpo="data/";
    string device="gpu";
    string FDMScheme="CentralDifferenceO4Iso2D";
    string timeScheme="RK4";
    double dt=0.001;
    double T=10;
    double dtExpo=1;
    int    NGrid=128;
    double L=NGrid*1.0;       
    
    // Generating a new system.
    System mySys(direExpo);
    // Generating a new mesh.
    Mesh mesh(2);
    mesh.setGridNumber(NGrid,NGrid,1);
    mesh.setBoxSize(L,L,1);
    // Add mesh to system.
    mySys.mesh_ptr=&mesh;
    
    // Creating fields.
    Field f(&mesh, "f",0);    
    Field phi(&mesh, "phi",1);
    Field psi(&mesh, "psi",1);
    Field aaa(&mesh, "aaa",1);
    
    f.setRhsTerms({
        {-1,{{&f}}}
    });
    
    phi.setRhsTerms({
        {{{"sin",&f,{2,0}}}}
    });

    psi.setRhsTerms({
        {{{"laplace",&f}}}
    });

    // FFuncFieldAddiPtrs fffap(&psi);
    aaa.setRhsTerms({
        {{{"atan2F",&f,{},{&psi}}}}
    });
    
    f.initFieldGaus(L/2, 0.1*L, 0.1);
    
    // phi.initFieldGaus(L/2, 0.1*L, 1.5);
    mySys.addField(&f);
    mySys.addField(&phi);
    mySys.addField(&psi);
    mySys.addField(&aaa);

    if (aaa.rhs_terms[0].f_funcs[0].f_func_fa_ptrs.f2_ptr != NULL) {
      cout <<"ptr="<<aaa.rhs_terms[0].f_funcs[0].f_func_fa_ptrs.f2_ptr;
      cout <<", name="<<(*aaa.rhs_terms[0].f_funcs[0].f_func_fa_ptrs.f2_ptr).name()<<endl;
    };
    if (phi.rhs_terms[0].f_funcs[0].f_func_fa_ptrs.f2_ptr != NULL) {
      cout <<"ptr="<<phi.rhs_terms[0].f_funcs[0].f_func_fa_ptrs.f2_ptr;
      cout <<", name="<<(*phi.rhs_terms[0].f_funcs[0].f_func_fa_ptrs.f2_ptr).name()<<endl;
    };    
    
    // Print system information.
    mySys.printSysInfo();
    
    // Creating an evolver:    
    Evolver evolver(&mySys,0,T,dt,dtExpo,device,timeScheme,FDMScheme);
    // evolver.initEvolver();
    // Running simulations
    evolver.run();
    
    // -----------------------------------------------------------
    
    return 0;
};

